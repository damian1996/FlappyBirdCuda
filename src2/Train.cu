#include "hip/hip_runtime.h"
#include "consts.h"
#include <thrust/device_vector.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
// https://stackoverflow.com/questions/8012304/why-defining-class-headers-without-cuda-device-attribute-works-c

__constant__ float FIRST_TREE = 400.0;
__constant__ float SECOND_TREE = 650.0;
__constant__ float THIRD_TREE = 900.0;



extern "C" {
	__device__
	void randInt(float& toRand, int interval) {
				int thid = blockIdx.x*blockDim.x+threadIdx.x;
				hiprandState state;
				hiprand_init(666, thid, 0, &state);
				toRand = hiprand(&state)%interval;
	 }
}

extern "C" {
	__device__
	void randFloat(float& toRand, int interval) {
				int thid = blockIdx.x*blockDim.x+threadIdx.x;
				hiprandState state;
				hiprand_init(666, thid, 0, &state);
				toRand = hiprand(&state)%interval;
	 }
}

struct Bird {
public:
  float x, y;
  bool active, moveUp;
	__device__
  Bird() {
    x = 100.0;
    y = (float) SCREEN_H/2;
    active = true;
    moveUp = false;
  }
	__device__
	void backInitialValues() {
    x = 100.0;
    y = (float) SCREEN_H/2;
    active = true;
    moveUp = false;
  }
};

struct Tree {
public:
  float x, h1, h2;
  bool add;
	__device__
	Tree() {}
  Tree(int coX) : x(coX){
    add = false;
  }
	__device__
  void heights() {
		h1 = 0.0;
		randInt(h1, 150);
    h1 = h1 + 100.0;
    h2 = h1 + DIFF_BETWEEN_H;
  }
};

struct List {
	Tree trees[5];
	int el = 0;
	int size = 0;
	__device__
	void addTree(Tree t) {
		trees[el].x = t.x;
		trees[el].h1 = t.h1;
		trees[el].h2 = t.h2;
		trees[el].add = t.add;
		el++;
		size += 1;
	}
	__device__
	void removeFromStart() {
		for(int i=1; i<size; i++) {
			trees[i-1] = trees[i]; // czy aby na pewno to dziala jak nalezy?
			el--;
			size--;
		}
	}
};

extern "C" {
	__device__
	 void fitness_function(int idx, float res) { // return float

	 }
}

extern "C" {
	__device__
   	void Mutation() {

		}
}

extern "C" {
	__device__
	void droppedTooLow(Bird& bird, bool r) { // bool
	  if(bird.y>=SCREEN_H) r = false;
	  else r = true;
	}
}

extern "C" {
	__device__
	void intersects(Bird& b, float CircX, float CircY, bool r) { // bool
	  float DeltaX = CircX - max(b.x, min(CircX, b.x + GIL_SIZE));
	  float DeltaY = CircY - max(b.y, min(CircY, b.y + GIL_SIZE));
		r = (DeltaX*DeltaX + DeltaY*DeltaY) < (CIRC_RAD*CIRC_RAD);
	}
}

extern "C" {
	__device__
		void collisionCheck(List& list, Bird& b, bool r) { // bool
			bool res;
			int siz = list.size;
			for(int i=0; i<siz; i++) {
	      float wspX = b.x + GIL_SIZE;
	      float wspY = b.y + GIL_SIZE;
	      if(list.trees[i].x<wspX && list.trees[i].x+WIDTH_TREE>b.x && list.trees[i].h1>b.y) { // && 0<wspY
					r = false;
					return;
				}
	      if(list.trees[i].x<wspX && list.trees[i].x+WIDTH_TREE>b.x && list.trees[i].h2<wspY) { // && SCREEN_H>gil[b].y
					r = false;
					return;
				}

	      float middleX = list.trees[i].x+WIDTH_TREE/2;
	      float middleY = list.trees[i].h1+(2*CIRC_RAD)/3;
				intersects(b, middleX, middleY, res);
	      if(res) {
					r = false;
					return;
				}
				middleY = list.trees[i].h2-(2*CIRC_RAD)/3;
	      intersects(b, middleX, middleY, res);
	      if(res) {
					r = false;
					return;
				}
	    }
	    r = true;
		}
}

extern "C" {
	__device__
		void getIBirdPosition(Bird& b, float wspX, float wspY) { // pair<float, float>
			wspX = b.x;
			wspY = b.y;
		}
}

extern "C" {
	__device__
		void getTreePosition(List& list, int i, Tree& t) {
			t.x = list.trees[i].x;
			t.h1 = list.trees[i].h1;
			t.h2 = list.trees[i].h2;
			// add?
		}
}

extern "C" {
	__device__
		void getNumberOfTrees(List& list, int siz) {
		   siz = list.size;
		}
}

extern "C" {
	__device__
	void update(Bird& b) {
	  if(b.moveUp) {
	    b.y -= BIRD_IMPR;
	  } else {
	    float decr = SCREEN_H/(FPS*TIME_FALL_BIRD);
	    b.y += decr;
	  }
	}
}

extern "C" {
	__device__
		void moveUp(Bird b) {
		  b.moveUp = true;
		}
}

extern "C" {
	__device__
	void createTree(List& list ,float i) {
		{
			Tree t;
			t.x = i;
		  t.heights();
		  list.addTree(t);
		}
	}
}

extern "C" {
	__device__
		void treeToRemove(List& list, Bird& b, bool r, int gameScore)
		{
        if(!list.trees[0].add && b.x>list.trees[0].x+WIDTH_TREE)
        {
           list.trees[0].add = true;
           gameScore++;
           r = true;
        }
				else
					 r = false;
		}
}

extern "C" {
	__device__
		void sigmoid(float hor, float hei, float& res) {
		  /*
			for(int i=0; i<9; i++) values[i] = 0;
		  values[0] = 1/(1 + exp(-hor-bias));
		  values[1] = 1/(1 + exp(-hei-bias));
		  for(int i=0; i<2; i++){
		    for(int j=0; j<6; j++){
		      values[j+2] += weights[i][j]*values[i];
		    }
		  }
		  for(int i=2; i<8; i++) {
		    values[i] = 1/(1 + exp(-values[i]-bias));
		    values[8] += weights[i][0]*values[i];
		  }
		  values[8] = 1/(1 + exp(-values[8]-bias));
		  return values[8];
			*/
		}
}

extern "C" {
	__global__
   	void TrainBirds(float* weights, float* fitness) {
			int gameScore = 0;
			List list;
			createTree(list, FIRST_TREE);
			createTree(list, SECOND_TREE);
			createTree(list, THIRD_TREE);
			Bird bird;
		}
}

				// vector drzew + inicjalizacja (3 poczatkowe drzewka oraz ptaszek)
				/*while(1)
				{
						treeToRemove();

						float decr = SCREEN_H/(FPS*TIME_FALL_BIRD);
						//for(auto& it : list_trees) // przesuniecie drzew
							//it.x -= MOVE_PIXS_ON_SEC/FPS;
						if(SCREEN_W - list_trees[list_trees.size()-1].x >= DIST_BET_TWO) {
							 createTree(list_trees[list_trees.size()-1].x + DIST_BET_TWO);
						}
						for(int i=0; i<NUMBER_DISPLAY_BOTS; i++) {
							if(gil[i].active) {
									int idx = list_trees[0].x > gil[i].x ? 0 : 1;
									float horizontal = list_trees[idx].x+WIDTH_TREE-gil[i].x;
									float heighDiff = ((list_trees[idx].y + list_trees[idx].height2)/2) - gil[i].y;
									float val = botToTrain.visibleNets[i].sigmoid(horizontal, heighDiff);
									if(val>=0) gil[i].moveUp = true;
									else gil[i].moveUp = false;
									//moveUp(i); // 0.5
									update(i);
									// to samo co na cudzie
							}
						}
						for(int i=0; i<NUMBER_DISPLAY_BOTS; i++) {
							//if(gil[i].active)
							 gil[i].active = collisionCheck(i);
							//printf("aha.. %d\n", gil[i].active);
						}

						for(int i=0; i<NUMBER_DISPLAY_BOTS; i++)
						{
							if(gil[i].active==true)
								gil[i].active = droppedTooLow(i);
						}
								if(allDead()==false) {
										//printf("Huh?");
										if(list_trees[0].x + WIDTH_TREE < 0)
											 list_trees.erase(list_trees.begin());

										for(int i=0; i<NUMBER_DISPLAY_BOTS; i++) {
												if(gil[i].active) {
													 update(i);
												}
										}
								}
								else if(allDead()==true){
									break;
								}
						}*/
