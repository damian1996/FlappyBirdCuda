#include "hip/hip_runtime.h"
#include "consts.h"
#include <thrust/device_vector.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
// https://stackoverflow.com/questions/8012304/why-defining-class-headers-without-cuda-device-attribute-works-c

extern "C" {
	__global__
	void Init(hiprandState *states) {
		int thid = blockIdx.x*blockDim.x+threadIdx.x;
		hiprand_init(clock64(), thid, 0, &states[thid]);
		__syncthreads();
	}
}

extern "C" {
	__device__
	void randInt(float& toRand, int interval, hiprandState* state) {
				int thid = blockIdx.x*blockDim.x+threadIdx.x;
				int tmp = hiprand(state);// %interval;
				if(tmp<0) tmp = -tmp;
				toRand = (tmp)%interval;
	 }
}

extern "C" {
	__device__
	void randFloat(float& toRand, float l, float r, hiprandState* state) {
				int thid = blockIdx.x*blockDim.x+threadIdx.x;
				int rand_max = 32767;
				float randomVarb, tmp;
				randInt(randomVarb, rand_max, state);
				toRand =  ((r - l) * (randomVarb / rand_max)) + l;
	 }
}

struct Bird {
public:
  float x, y;
  bool active, moveUp;
	__device__
  Bird() {
    x = 100.0;
    y = (float) SCREEN_H/2;
    active = true;
    moveUp = false;
  }
	__device__
	void backInitialValues() {
    x = 100.0;
    y = (float) SCREEN_H/2;
    active = true;
    moveUp = false;
  }
};

struct Tree {
public:
  float x, h1, h2;
  bool add;
	__device__
	Tree() {}
  Tree(int coX) : x(coX){
    add = false;
  }
	__device__
  void heights(hiprandState* state) {
		h1 = 0.0;
		randInt(h1, 150, state);
    h1 = h1 + 100.0;
    h2 = h1 + DIFF_BETWEEN_H;
  }
};

struct List {
public:
	Tree trees[5];
	int el, size;
	__device__
	List() {
		el = 0;
		size = 0;
	}
	__device__
	void addTree(Tree t) {
		trees[el].x = t.x;
		trees[el].h1 = t.h1;
		trees[el].h2 = t.h2;
		trees[el].add = t.add;
		el++;
		size += 1;
	}
	__device__
	void removeFromStart() {
		for(int i=1; i<size; i++) {
			trees[i-1].x = trees[i].x; // czy aby na pewno to dziala jak nalezy?
			trees[i-1].h1 = trees[i].h1;
			trees[i-1].h2 = trees[i].h2;
			trees[i-1].add = trees[i].add;
			el--;
			size--;
		}
	}
};

extern "C" {
	__device__
		void fitness_function(List& list, int gameScore, Bird& bird, float res) { // return float
				 int idx = list.trees[0].x > bird.x ? 0 : 1;
				 res = DIST_BET_TWO*gameScore + 200 + 250 - (list.trees[idx].x - bird.x);
		// fitness = total travelled distance - distance to the closest gap
	 }
}

extern "C" {
	__device__
	void droppedTooLow(Bird& bird) { // bool
	  if(bird.y>=SCREEN_H) bird.active = false;
	  else bird.active = true;
	}
}

extern "C" {
	__device__
	void intersects(Bird& b, float CircX, float CircY, bool r) { // bool
	  float DeltaX = CircX - max(b.x, min(CircX, b.x + GIL_SIZE));
	  float DeltaY = CircY - max(b.y, min(CircY, b.y + GIL_SIZE));
		r = (DeltaX*DeltaX + DeltaY*DeltaY) < (CIRC_RAD*CIRC_RAD);
	}
}

extern "C" {
	__device__
		void collisionCheck(List& list, Bird& b) { // bool
			bool res;
			int siz = list.size;
			for(int i=0; i<siz; i++) {
	      float wspX = b.x + GIL_SIZE;
	      float wspY = b.y + GIL_SIZE;
	      if(list.trees[i].x<wspX && list.trees[i].x+WIDTH_TREE>b.x && list.trees[i].h1>b.y) { // && 0<wspY
					b.active = false;
					return;
				}
	      if(list.trees[i].x<wspX && list.trees[i].x+WIDTH_TREE>b.x && list.trees[i].h2<wspY) { // && SCREEN_H>gil[b].y
					b.active = false;
					return;
				}

	      float middleX = list.trees[i].x+WIDTH_TREE/2;
	      float middleY = list.trees[i].h1+(2*CIRC_RAD)/3;
				intersects(b, middleX, middleY, res);
	      if(res) {
					b.active = false;
					return;
				}
				middleY = list.trees[i].h2-(2*CIRC_RAD)/3;
	      intersects(b, middleX, middleY, res);
	      if(res) {
					b.active = false;
					return;
				}
	    }
	    b.active = true;
		}
}

extern "C" {
	__device__
		void getIBirdPosition(Bird& b, float wspX, float wspY) { // pair<float, float>
			wspX = b.x;
			wspY = b.y;
		}
}

extern "C" {
	__device__
		void getTreePosition(List& list, int i, Tree& t) {
			t.x = list.trees[i].x;
			t.h1 = list.trees[i].h1;
			t.h2 = list.trees[i].h2;
			t.add = list.trees[i].add;
		}
}

extern "C" {
	__device__
		void getNumberOfTrees(List& list, int siz) {
		   siz = list.size;
		}
}

extern "C" {
	__device__
	void update(Bird& b) {
	  if(b.moveUp) {
	    b.y -= BIRD_IMPR;
	  } else {
	    float decr = SCREEN_H/(FPS*TIME_FALL_BIRD);
	    b.y += decr;
	  }
	}
}

extern "C" {
	__device__
	void createTree(List& list, float i, hiprandState* state) {
		{
			Tree t;
			t.x = i;
		  t.heights(state);
		  list.addTree(t);
		}
	}
}

extern "C" {
	__device__
		void treeToRemove(List& list, Bird& b, int gameScore)
		{
        if(!list.trees[0].add && b.x>list.trees[0].x+WIDTH_TREE)
        {
           list.trees[0].add = true;
           gameScore++;
        }
		}
}

extern "C" {
	__device__
		void sigmoid(float* weights, float hor, float hei, float& res) {
			int thid = threadIdx.x + blockIdx.x*blockDim.x;
			float values[9];
			int offset = thid*WEIGHTS_SIZE;
			for(int i=0; i<9; i++) values[i] = 0;
		  values[0] = 1/(1 + exp(-hor-1));
		  values[1] = 1/(1 + exp(-hei-1));
		  for(int i=0; i<2; i++) {
		    for(int j=0; j<6; j++) {
		      values[j+2] += weights[offset+i*6+j]*values[i];
		    }
		  }
		  for(int i=2; i<8; i++) {
		    values[i] = 1/(1 + exp(-values[i]-1));
		    values[8] +=  weights[offset+i*6+0]*values[i];
		  }
			res = 1/(1 + exp(-values[8]-0.5));
		}
}

extern "C" {
	__global__
   	void TrainBirds(float* weights, float* fitness, hiprandState* states) {
			int thid = blockIdx.x*blockDim.x+threadIdx.x;
			int gameScore = 0;
			List list;
			createTree(list, FIRST_TREE, &states[thid]);
			createTree(list, SECOND_TREE, &states[thid]);
			createTree(list, THIRD_TREE, &states[thid]);
			Bird bird;
			while(1) {
				treeToRemove(list, bird, gameScore);
				float decr = SCREEN_H/(FPS*TIME_FALL_BIRD);
				for(int i=0; i<list.size; i++)
					list.trees[i].x -= MOVE_PIXS_ON_SEC/FPS;
				if(SCREEN_W - list.trees[list.size-1].x >= DIST_BET_TWO) {
					 createTree(list, list.trees[list.size-1].x + DIST_BET_TWO, &states[thid]);
				}
				if(bird.active) {
					int idx = list.trees[0].x > bird.x ? 0 : 1;
					float horizontal = list.trees[idx].x + WIDTH_TREE - bird.x;
					float heighDiff = ((list.trees[idx].h1 + list.trees[idx].h2)/2) - bird.y;
					float val = 0.0;
				  sigmoid(weights, horizontal, heighDiff, val);
					//if(thid<10) {
						//printf("%d %f\n", thid, val);
				  //}
					if(val>0.5) bird.moveUp = true;
					else bird.moveUp = false;
				}
				collisionCheck(list, bird);
				if(bird.active)
					droppedTooLow(bird);
				if(bird.active) {
						if(list.trees[0].x + WIDTH_TREE < 0) {
							list.removeFromStart();
						}
						if(bird.active)
							update(bird);
				} else {
					break;
				}
			}
			float res;
			fitness_function(list, gameScore, bird, res);
			fitness[thid] = res;
			__syncthreads();
	 }
}

extern "C" {
	__global__
   	void Mutation(float* weights, float* crosses, hiprandState *states) {
				int thid = blockDim.x*blockIdx.x + threadIdx.x;
				int offset = thid*WEIGHTS_SIZE;
				int ms = thid/NMB_CROSSES;
				int off_crs = ms*WEIGHTS_SIZE;
				for(int i=0; i<9; i++) {
						for(int j=0; j<6; j++) {
							float tmp, prev = crosses[off_crs+i*6+j];
							randFloat(tmp, prev-EPS, prev+EPS, &states[thid]);
							weights[offset + i*6 + j] = tmp;
						}
				}
				__syncthreads();
		}
}
