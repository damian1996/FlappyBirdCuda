#include "hip/hip_runtime.h"
#include "consts.h"
#include <hiprand/hiprand_kernel.h>
#include <cmath>

extern "C" {
	__global__
	void Init(hiprandState *states) {
		int thid = blockIdx.x*blockDim.x+threadIdx.x;
		hiprand_init(clock64(), thid, 0, &states[thid]);
		__syncthreads();
	}
}

extern "C" {
	__device__
	void randInt(float& toRand, int interval, hiprandState* state) {
				int tmp = hiprand(state);
				if(tmp<0) tmp = -tmp;
				toRand = (tmp)%interval;
	 }
}

extern "C" {
	__device__
	void randFloat(float& toRand, float l, float r, hiprandState* state) {
				int rand_max = 32767;
				float randomVarb;
				randInt(randomVarb, rand_max, state);
				toRand =  ((r - l) * (randomVarb / rand_max)) + l;
	 }
}

struct Bird {
public:
  float x, y, fit, acc, speed;
  bool active, moveUp;
	__device__
  Bird() {
    x = INITIAL_POSITION_BIRD_X;
    y = (float) SCREEN_H/2;
		fit = 0;
		speed = 0;
		acc = 550;
    active = true;
    moveUp = false;
  }
	__device__
	void backInitialValues() {
		x = INITIAL_POSITION_BIRD_X;
		y = (float) SCREEN_H/2;
		fit = 0;
		speed = 0;
		acc = 550;
		active = true;
		moveUp = false;
  }
};

struct Tree {
public:
  float x, h1, h2;
  bool scored_tree;
	__device__
	Tree() {}
  Tree(int x) : x(x){

  }
	__device__
  void heights(hiprandState* state) {
		h1 = 0.0;
		randInt(h1, RANGE_RAND_DIST_TREES, state);
    h1 = h1 + MIN_HEIGHT_FIRST_TREE;
    h2 = h1 + DIFF_BETWEEN_H;
		scored_tree = false;
  }
};

struct List {
public:
	Tree trees[6];
	int size;
	__device__
	List() {
		size = 0;
	}
	__device__
	void addTree(Tree t) {
		trees[size].x = t.x;
		trees[size].h1 = t.h1;
		trees[size].h2 = t.h2;
		trees[size++].scored_tree = t.scored_tree;
	}
	__device__
	void removeFromStart() {
		for(int i=1; i<size; i++) {
			trees[i-1].x = trees[i].x;
			trees[i-1].h1 = trees[i].h1;
			trees[i-1].h2 = trees[i].h2;
			trees[i-1].scored_tree = trees[i].scored_tree;
		}
		size--;
	}
};

extern "C" {
	__device__
	void createTree(List& list, float i, hiprandState* state) {
		{
			int thid = blockIdx.x*blockDim.x+threadIdx.x;
			Tree t;
			t.x = i;
		  t.heights(state);
		  list.addTree(t);
		}
	}
}

extern "C" {
	__device__
		void treeToRemove(List& list, Bird& b, int gameScore)
		{
        if(!list.trees[0].scored_tree && b.x>(list.trees[0].x+WIDTH_TREE))
        {
           list.trees[0].scored_tree = true;
           gameScore++;
        }
		}
}

extern "C" {
	__device__
	void update(Bird& b) {
		  float t = 1.0/FPS;
		  if(b.moveUp) {
		    	b.y += b.speed*t;
		   		b.speed = -180;
		  } else {
		    	b.y += b.speed*t;
		   		b.speed += b.acc*t;
		  }
	}
}

extern "C" {
	__device__
		void sigmoid(float* weights, float hor, float hei, float speed, float* res) {
			int thid = blockIdx.x*blockDim.x+threadIdx.x;
			float values[wx];
			int offset = thid*WEIGHTS_SIZE;
			for(int i=0; i<wx; i++) values[i] = 0;
			values[0] = 1/(1 + exp(-hor));
		  values[1] = 1/(1 + exp(-hei));
			values[2] = 1/(1 + exp(-speed));
			for(int i=0; i<inpVals; i++) {
		    for(int j=0; j<wy; j++) {
		      values[j+inpVals] += (weights[offset + i*6 + j]*values[i]);
		    }
		  }
			for(int i=inpVals; i<wx-1; i++) {
				values[i] = 1/(1 + exp(-values[i]));
				values[wx-1] +=  weights[offset+i*wy+0]*values[i];
			}
			values[wx-1] = 1/(1 + exp(-values[wx-1]));
			*res = values[wx-1];
		}
}

extern "C" {
	__device__
	void intersects(Bird& b, float CircX, float CircY, bool& r) {
	  float DeltaX = CircX - max(b.x, min(CircX, b.x + GIL_SIZE));
	  float DeltaY = CircY - max(b.y, min(CircY, b.y + GIL_SIZE));
		r = (DeltaX*DeltaX + DeltaY*DeltaY) < (CIRC_RAD*CIRC_RAD);
	}
}

extern "C" {
	__device__
		void collisionCheck(List& list, Bird& b) {
				bool res;
				int siz = list.size;
				for(int i=0; i<siz; i++) {
			      float wspX = b.x + GIL_SIZE;
			      float wspY = b.y + GIL_SIZE;
			      if(list.trees[i].x<wspX && list.trees[i].x+WIDTH_TREE>b.x && list.trees[i].h1>b.y) {
								b.active = false;
								return;
						}
			      if(list.trees[i].x<wspX && list.trees[i].x+WIDTH_TREE>b.x && list.trees[i].h2<wspY) {
								b.active = false;
								return;
						}

			      float middleX = list.trees[i].x+WIDTH_TREE/2;
			      float middleY = list.trees[i].h1+(2*CIRC_RAD)/3;
						intersects(b, middleX, middleY, res);
			      if(res) {
								b.active = false;
								return;
						}
						middleY = list.trees[i].h2-(2*CIRC_RAD)/3;
			      intersects(b, middleX, middleY, res);
			      if(res) {
								b.active = false;
								return;
						}
		    }
		    b.active = true;
		}
}

extern "C" {
	__device__
	void droppedTooLow(Bird& bird) {
	  if(bird.y>=SCREEN_H) bird.active = false;
	  else bird.active = true;
	}
}

extern "C" {
	__global__
   	void TrainBirds(float* weights, float* biases, float* fitness, hiprandState* states) {
			int thid = blockIdx.x*blockDim.x+threadIdx.x;
			int gameScore = 0;
			List list;
			createTree(list, FIRST_TREE, &states[thid]);
			createTree(list, SECOND_TREE, &states[thid]);
			createTree(list, THIRD_TREE, &states[thid]);
			Bird bird;
			while(1) {
				treeToRemove(list, bird, gameScore);
				for(int i=0; i<list.size; i++)
					list.trees[i].x -= MOVE_PIXS_ON_SEC/FPS;

				if(list.trees[0].x + WIDTH_TREE < 0) {
					list.removeFromStart();
				}

				if(SCREEN_W - list.trees[list.size-1].x >= DIST_BET_TWO) {
					 createTree(list, list.trees[list.size-1].x + DIST_BET_TWO, &states[thid]);
				}

				if(bird.active) {
						int idx = ((list.trees[0].x + WIDTH_TREE) > bird.x) ? 0 : 1;
						float horizontal = list.trees[idx].x + WIDTH_TREE - bird.x;
						float heighDiff = ((list.trees[idx].h1 + list.trees[idx].h2)/2) - bird.y;
						float sp = bird.speed;
						float val = 0.0;
					  sigmoid(weights, horizontal, heighDiff, sp, &val);
						if(val>biases[thid]) bird.moveUp = true;
						else bird.moveUp = false;
						update(bird);
						collisionCheck(list, bird);
				}
				if(bird.active) {
					droppedTooLow(bird);
				}

				if(!bird.active) {
					int idx = list.trees[0].x+WIDTH_TREE > bird.x ? 0 : 1;
					float heighDiff = ((list.trees[idx].h1 + list.trees[idx].h2)/2) - bird.y;
					if(heighDiff<0) heighDiff = -heighDiff;
					bird.fit += -heighDiff;
					break;
				} else {
					float end = (MOVE_PIXS_ON_SEC/FPS)*250;
					bird.fit += MOVE_PIXS_ON_SEC/FPS;
					if(bird.fit > end) break;
				}
			}
			fitness[thid] = bird.fit;
			__syncthreads();
	 }
}

extern "C" {
	__global__
	void Mutation(float* weights, float* biases, hiprandState *states) {
		int thid = blockDim.x*blockIdx.x + threadIdx.x;
		int offset = thid*WEIGHTS_SIZE;
		int divider = UNITY_IN_TRAINING/CROSSOVER_BIRDS;
		int block = thid/divider;
		int main_id = block*divider;
		if((thid%divider)>0) {
				for(int i=0; i<wx; i++) {
						for(int j=0; j<wy; j++) {
								float tmp = 0;
								float prev = weights[main_id*WEIGHTS_SIZE + i*wy + j];
								randFloat(tmp, prev-EPS, prev+EPS, &states[thid]);
								weights[offset + i*wy + j] = tmp;
						}
				}
				float temp;
				randFloat(temp, biases[main_id]-0.05, biases[main_id]+0.05, &states[thid]);
				biases[thid] = temp;
		}
		__syncthreads();
	}
}
